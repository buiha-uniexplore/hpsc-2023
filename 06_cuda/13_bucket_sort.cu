
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void initiate(int *bucket){
  bucket[threadIdx.x] = 0;
}

__global__ void increment(int *bucket, int *key){

  int i = key[threadIdx.x];
 
  atomicAdd(&bucket[i], 1);

}

__global__ void offsetcalc(int *a, int *b, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j=1; j<n; j<<=1){
    b[i] = a[i];
    __syncthreads();
    a[i] += b[i-j];
    __syncthreads();
  }

}

__global__ void sort(int *bucket, int *key, int *offset){
  int i = threadIdx.x;

  int offsetnum = offset[i-1];

  for (int jj=0; jj<bucket[i]; jj++){
    key[jj+offsetnum] = i;
  }

}


int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  // First for loop
  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  initiate<<<1,range>>>(bucket);
  hipDeviceSynchronize();

  // Second for loop
  increment<<<1,n>>>(bucket, key);
  hipDeviceSynchronize();

  // Calculating offset for third loop
  int *offset, *offsetout;
  hipMallocManaged(&offset, range*sizeof(int));
  hipMallocManaged(&offsetout, range*sizeof(int));

  for (int i = 0; i<range; i++){
    offset[i] = bucket[i];
  }

  offsetcalc<<<1, range>>>(offset, offsetout, range);
  hipDeviceSynchronize();

  // Third for loop
  sort<<<1,range>>>(bucket, key, offset);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(bucket);
  hipFree(key);
  hipFree(offset);
  hipFree(offsetout);
}
