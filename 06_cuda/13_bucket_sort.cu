
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void initiate(int *bucket){
  bucket[threadIdx.x] = 0;
}

__global__ void increment(int *bucket, int *key){

  int i = key[threadIdx.x];
 
  atomicAdd(&bucket[i], 1);

}

__global__ void sort(int * bucket, int *key){
  int i = threadIdx.x;
  int offset = 0;
  for (int k=0; k<i; k++) {
    offset += bucket[k];
  }
  for (int jj=0; jj<bucket[i]; jj++){
    key[jj+offset] = i;
  }

}


int main() {
  int n = 50;
  int range = 5;
  //std::vector<int> key(n);
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  /*
  std::vector<int> bucket(range); 
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  */

  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  initiate<<<1,range>>>(bucket);
  hipDeviceSynchronize();
  for (int i=0; i<range; i++) {
  //  printf("%d ", bucket[i]);
  }
  //printf("\n");

  /*
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }
  */

  increment<<<1,n>>>(bucket, key);
  hipDeviceSynchronize();
  for (int i=0; i<range; i++) {
  //  printf("%d ", bucket[i]);
  }
  //printf("\n");

  // for (int i=0, j=0; i<range; i++) {
  //   for (; bucket[i]>0; bucket[i]--) {
  //     key[j++] = i;
  //   }
  // }

  sort<<<1,range>>>(bucket, key);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(bucket);
}
