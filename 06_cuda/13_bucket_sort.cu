#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void initiate(int *bucket){
  bucket[threadIdx.x] = 0;
}

__global__ void increment(int *bucket, int *key){

  int i = key[threadIdx.x];
 
  atomicAdd(&bucket[i], 1);

}

__device__ __managed__ int jj=0;

__global__ void sort(int * bucket, int *key, int &jj){
  i = threadIdx.x;
  extern __shared__ int jjj;
  for (int t = bucket[i]; bucket[i] < 0; bucket[i]--){
    
  }

}


int main() {
  int n = 50;
  int range = 5;
  //std::vector<int> key(n);
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  /*
  std::vector<int> bucket(range); 
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  */

  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  initiate<<<1,range>>>(bucket);
  hipDeviceSynchronize();
  for (int i=0; i<range; i++) {
    printf("%d ", bucket[i]);
  }
  printf("\n");

  /*
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }
  */

  increment<<<1,n>>>(bucket, key);
  hipDeviceSynchronize();
  for (int i=0; i<range; i++) {
    printf("%d ", bucket[i]);
  }
  printf("\n");

  // for (int i=0, j=0; i<range; i++) {
  //   for (; bucket[i]>0; bucket[i]--) {
  //     key[j++] = i;
  //   }
  // }

  sort<<<1,range>>>(bucket, key, jj);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(bucket);
}
