
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void initiate(int *bucket){
  bucket[threadIdx.x] = 0;
}

__global__ void increment(int *bucket, int *key){

  int i = key[threadIdx.x];
 
  atomicAdd(&bucket[i], 1);

}

__global__ void offsetcalc(int *a, int *b, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j=1; j<n; j<<=1){
    b[i] = a[i];
    __syncthreads();
    a[i] += b[i-j];
    __syncthreads();
  }
  for (int i=0; i<n; i++) {
  //  printf("%d ",a[i]);
  }
}

__global__ void sort(int *bucket, int *key, int *offset){
  int i = threadIdx.x;

  int offsetnum = offset[i-1];

  for (int jj=0; jj<bucket[i]; jj++){
    key[jj+offsetnum] = i;
  }

}


int main() {
  int n = 50;
  int range = 5;
  //std::vector<int> key(n);
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  /*
  std::vector<int> bucket(range); 
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  */

  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  initiate<<<1,range>>>(bucket);
  hipDeviceSynchronize();
  for (int i=0; i<range; i++) {
  //  printf("%d ", bucket[i]);
  }
  //printf("\n");

  /*
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }
  */

  increment<<<1,n>>>(bucket, key);
  hipDeviceSynchronize();
  for (int i=0; i<range; i++) {
    printf("%d ", bucket[i]);
  }
  printf("\n");

  // for (int i=0, j=0; i<range; i++) {
  //   for (; bucket[i]>0; bucket[i]--) {
  //     key[j++] = i;
  //   }
  // }

  int *offset, *offsetout;
  hipMallocManaged(&offset, range*sizeof(int));
  hipMallocManaged(&offsetout, range*sizeof(int));

  for (int i = 0; i<range; i++){
    offset[i] = bucket[i];
  }


  offsetcalc<<<1, range>>>(offset, offsetout, range);
  hipDeviceSynchronize();
  for (int i=0; i<range; i++) {
    printf("%d ",offsetout[i]);
  }
  printf("\n");

  sort<<<1,range>>>(bucket, key, offset);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(bucket);
  hipFree(key);
  hipFree(offset);
  hipFree(offsetout);
}
